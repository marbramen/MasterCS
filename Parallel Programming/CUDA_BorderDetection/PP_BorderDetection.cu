#include "hip/hip_runtime.h"
/*
 * César Bragagnini
 * Detección de Bordes de forma serial, paralela usando CUDA
 * Curso: Programación Paralela
 * Se usa OpenCV para abrir y guardar las imagenes
 */
#include "opencv2/highgui/highgui.hpp"
#include <cstdio>
#include <time.h>

using namespace cv;

#define MAX_THREADS_BY_BLOCK 1024
#define DIM_BLOCK_X 32
#define DIM_BLOCK_Y 32
#define DIM_GRID_Y 10

int nRows, nCols;
inline int cmpByMaxUmbral(int x){ return x > 255 ? 255 : x; }
inline int myAbs(int x){ return x > 0 ? x : -1 * x;}
inline int gPos(int y, int x){ return y * nCols + x; }

/*
 * Se aplica filtro : X_HORIZONTAL, Y_VERTICAL
 * SobelX: SobX[3][3] = {{-1,0,1},{-2,0,2},{-1,0,2}}
 * SobelY: SobY[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}}
*/


void serialSobelAsImageIn1d(Mat inMatImage){
	Mat matImgFinal = inMatImage.clone();
	nRows = inMatImage.rows;
	nCols = inMatImage.cols;

	/*
	 * Considerar que la image se guarda en una matriz 1d
	 * se debe buscar la posicion correcta
	 */
	int* inImage = new int[nRows * nCols];
	for(int y = 0; y < nRows; y++)
		for(int x = 0; x < nCols; x++)
			inImage[gPos(y, x)] = inMatImage.at<uchar>(y,x);


	int* resImage = new int[nRows * nCols];

	clock_t time = clock();
	for(int y = 1; y < nRows - 1; y++){
		for(int x = 1; x < nCols - 1; x++){
			int valueX = (inImage[gPos(y-1, x+1)] + 2 * inImage[gPos(y, x+1)] + inImage[gPos(y+1, x+1)]) - (inImage[gPos(y-1, x-1)] + 2 * inImage[gPos(y, x-1)] + inImage[gPos(y+1, x-1)]);
			int valueY = (inImage[gPos(y-1, x-1)] + 2 * inImage[gPos(y-1, x)] + inImage[gPos(y-1, x+1)]) - (inImage[gPos(y+1, x-1)] + 2 * inImage[gPos(y+1, x)] + inImage[gPos(y+1, x+1)]);
			resImage[gPos(y, x)] = cmpByMaxUmbral(myAbs(valueX) + myAbs(valueY));
		}
	}
	time = clock() - time;

	for(int y = 0; y < nRows; y++)
		for(int x = 0; x < nCols; x++)
			matImgFinal.at<uchar>(y,x) = resImage[gPos(y, x)];
	delete(inImage);
	delete(resImage);
	imwrite("/home/cesar/CesarBragagnini/MCS/others/resSerialImage1d.png", matImgFinal);
	printf("termino filtro 1d en %f s.\n", ((float)(time))/CLOCKS_PER_SEC);
}


void serialSobelAsImageIn2d(Mat inMatImage){
    Mat matImgFinal = inMatImage.clone();
	nRows = inMatImage.rows;
	nCols = inMatImage.cols;

	/*
	 * La imagen se guarda en una matriz 2d
	 */
	int **inImage = new int*[nRows];
	for(int y = 0; y < nRows; y++){
		inImage[y] = new int[nCols];
		for(int x = 0; x < nCols; x++){
			inImage[y][x] = inMatImage.at<uchar>(y,x);
		}
	}

	int **resImag = new int*[nRows];
	resImag[0] = new int[nCols];

	clock_t time = clock();
	for(int y = 1; y < nRows - 1; y++){
		resImag[y] = new int[nCols];
		for(int x = 1; x < nCols - 1; x++){
			int valueX = (inImage[y-1][x+1] + 2 * inImage[y][x+1] + inImage[y+1][x+1]) - (inImage[y-1][x-1] + 2 * inImage[y][x-1] + inImage[y+1][x-1]) ;
			int valueY = (inImage[y-1][x-1] + 2 * inImage[y-1][x] + inImage[y-1][x+1]) - (inImage[y+1][x-1] + 2 * inImage[y+1][x] + inImage[y+1][x+1]) ;
			resImag[y][x] = cmpByMaxUmbral(myAbs(valueX) + myAbs(valueY));
		}
	}
	time = clock() - time;

    for(int y = 1; y < nRows - 1; y++){
    	for(int x = 1; x < nCols - 1; x++){
    		matImgFinal.at<uchar>(y, x) = resImag[y][x];
    	}
    }
    delete(inImage);
	imwrite("/home/cesar/CesarBragagnini/MCS/others/resSerialImage2d.png", matImgFinal);
    printf("termino filtro en 2d en %f  s.\n", ((float)(time))/CLOCKS_PER_SEC);
}

__device__ int cuGPos(int y, int x, int cuCols){
	return y * cuCols + x;
}

__global__ void cuda1ThreadByPixelWithGrid2dBlock2d(int *cuPoRows, int *cuPoCols, int *inImage, int *resImage){

	int inx = threadIdx.x + blockIdx.x * blockDim.x;
	int iny = threadIdx.y + blockIdx.y * blockDim.y;
	int threadIdGlob = inx + iny * gridDim.x;

	int cuRows = *cuPoRows;
	int cuCols = *cuPoCols;

	if(threadIdGlob < cuRows * cuCols){
		int y = threadIdGlob / cuCols;
		int x = threadIdGlob % cuCols;
		/*
		 * Aplicamos operador Sobel
		 * Tenemos que buscar en el array1d los vecinos de X,y
		*/
		int valueX = (inImage[cuGPos(y-1,x+1,cuCols)] + 2 * inImage[cuGPos(y,x+1,cuCols)] + inImage[cuGPos(y+1,x+1,cuCols)]) - (inImage[cuGPos(y-1,x-1,cuCols)] + 2 * inImage[cuGPos(y,x-1,cuCols)] + inImage[cuGPos(y+1,x-1,cuCols)]);
		int valueY = (inImage[cuGPos(y-1,x-1,cuCols)] + 2 * inImage[cuGPos(y-1,x,cuCols)] + inImage[cuGPos(y-1,x+1,cuCols)]) - (inImage[cuGPos(y+1,x-1,cuCols)] + 2 * inImage[cuGPos(y+1,x,cuCols)] + inImage[cuGPos(y+1,x+1,cuCols)]);
		valueX = valueX > 0 ? valueX : -1 * valueX;
		valueY = valueY > 0 ? valueY : -1 * valueY;
		resImage[cuGPos(y,x,cuCols)] = valueX  + valueY > 255 ? 255 : valueX + valueY;
	 }
}


__global__ void cuda1ThreadByPixelWithGrid1dBlock1d(int *cuPoRows, int *cuPoCols, int *inImage, int *resImage){

	int threadIdGlob = threadIdx.x + blockIdx.x * blockDim.x ;

	int cuRows = *cuPoRows;
	int cuCols = *cuPoCols;

	if(threadIdGlob < cuRows * cuCols){
		int y = threadIdGlob / cuCols;
		int x = threadIdGlob % cuCols;
		/*
		 * Aplicamos operador Sobel
		 * Tenemos que buscar en el array1d los vecinos de X,y
		*/
//		int valueX = (inImage[gPos(y-1, x+1)] + 2 * inImage[gPos(y, x+1)] + inImage[gPos(y+1, x+1)]) - (inImage[gPos(y-1, x-1)] + 2 * inImage[gPos(y, x-1)] + inImage[gPos(y+1, x-1)]);
//		int valueY = (inImage[gPos(y-1, x-1)] + 2 * inImage[gPos(y-1, x)] + inImage[gPos(y-1, x+1)]) - (inImage[gPos(y+1, x-1)] + 2 * inImage[gPos(y+1, x)] + inImage[gPos(y+1, x+1)]);
		int valueX = (inImage[cuGPos(y-1,x+1,cuCols)] + 2 * inImage[cuGPos(y,x+1,cuCols)] + inImage[cuGPos(y+1,x+1,cuCols)]) - (inImage[cuGPos(y-1,x-1,cuCols)] + 2 * inImage[cuGPos(y,x-1,cuCols)] + inImage[cuGPos(y+1,x-1,cuCols)]);
		int valueY = (inImage[cuGPos(y-1,x-1,cuCols)] + 2 * inImage[cuGPos(y-1,x,cuCols)] + inImage[cuGPos(y-1,x+1,cuCols)]) - (inImage[cuGPos(y+1,x-1,cuCols)] + 2 * inImage[cuGPos(y+1,x,cuCols)] + inImage[cuGPos(y+1,x+1,cuCols)]);
		valueX = valueX > 0 ? valueX : -1 * valueX;
		valueY = valueY > 0 ? valueY : -1 * valueY;
		resImage[cuGPos(y,x,cuCols)] = valueX  + valueY > 255 ? 255 : valueX + valueY;
	 }
}


void cudaSobelInLinearMemoryGrid1dBlock1d(Mat inMatImage){
	Mat matImgFinal = inMatImage.clone();
	nRows = inMatImage.rows;
	nCols = inMatImage.cols;

	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	float time;

	int* inImage = new int[nRows * nCols];
	for(int y = 0; y < nRows; y++)
		for(int x = 0; x < nCols; x++)
			inImage[gPos(y, x)] = inMatImage.at<uchar>(y,x);

	int *cuPoRows, *cuPoCols, *cuInImage, *cuResImage;
	hipMalloc((void**) &cuPoRows, sizeof(int));
	hipMalloc((void**) &cuPoCols, sizeof(int));
	hipMalloc((void**) &cuInImage, nRows * nCols * sizeof(int));
	hipMalloc((void**) &cuResImage, nRows * nCols * sizeof(int));

	hipMemcpy(cuPoRows, &nRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuPoCols, &nCols, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuInImage, inImage, nRows * nCols * sizeof(int), hipMemcpyHostToDevice);

	int N = nRows * nCols;
	dim3 blockDim(MAX_THREADS_BY_BLOCK,1,1);
	dim3 gridDim((N + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK,1,1);

	hipEventRecord( start, 0 );
	cuda1ThreadByPixelWithGrid1dBlock1d<<<gridDim, blockDim>>>(cuPoRows, cuPoCols, cuInImage, cuResImage);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	hipMemcpy(inImage, cuResImage, nRows * nCols * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuPoRows);
	hipFree(cuPoCols);
	hipFree(cuInImage);
	hipFree(cuResImage);

	for(int y = 1; y < nRows - 1; y++)
		for(int x = 1; x < nCols - 1; x++)
			matImgFinal.at<uchar>(y, x) = inImage[gPos(y, x)];
	delete(inImage);

	imwrite("/home/cesar/CesarBragagnini/MCS/others/resCudaImageGr1dBl1d.png", matImgFinal);
	printf("termino cuda Grid1d Block1d en %f s.\n", time / 1000.0);
}


void cudaSobelInLinearMemoryGrid2dBlock2d(Mat inMatImage, int heightGrid){
	Mat matImgFinal = inMatImage.clone();
	nRows = inMatImage.rows;
	nCols = inMatImage.cols;

	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	float time;

	int* inImage = new int[nRows * nCols];
	for(int y = 0; y < nRows; y++)
		for(int x = 0; x < nCols; x++)
			inImage[gPos(y, x)] = inMatImage.at<uchar>(y,x);

	int *cuPoRows, *cuPoCols, *cuInImage, *cuResImage;
	hipMalloc((void**) &cuPoRows, sizeof(int));
	hipMalloc((void**) &cuPoCols, sizeof(int));
	hipMalloc((void**) &cuInImage, nRows * nCols * sizeof(int));
	hipMalloc((void**) &cuResImage, nRows * nCols * sizeof(int));

	hipMemcpy(cuPoRows, &nRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuPoCols, &nCols, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuInImage, inImage, nRows * nCols * sizeof(int), hipMemcpyHostToDevice);

	int N = nRows * nCols;
	int nBloq = (N + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK;

	dim3 blockDim(DIM_BLOCK_X,DIM_BLOCK_Y,1);
	dim3 gridDim((nBloq + heightGrid - 1) / heightGrid, heightGrid,1);

	hipEventRecord( start, 0 );
	cuda1ThreadByPixelWithGrid2dBlock2d<<<gridDim, blockDim>>>(cuPoRows, cuPoCols, cuInImage, cuResImage);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	hipMemcpy(inImage, cuResImage, nRows * nCols * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuPoRows);
	hipFree(cuPoCols);
	hipFree(cuInImage);
	hipFree(cuResImage);

	for(int y = 1; y < nRows - 1; y++)
		for(int x = 1; x < nCols - 1; x++)
			matImgFinal.at<uchar>(y, x) = inImage[gPos(y, x)];

	delete(inImage);
	imwrite("/home/cesar/CesarBragagnini/MCS/others/resCudaImageGr2dBl2d.png", matImgFinal);
	printf("termino cuda en Grid2d Block2d gridY:%d %f s.\n",heightGrid, time / 1000.0);
}

int main( )
{
	/*
	 * Se usa OpenCV para leer y guardar imagenes,
	 * OpenCV considera
	 * al eje Y de arriba hacia abajo
	 * al eje X de izquierda a derecha
	*/
	int width[3] = {10, 30, 50};
	Mat inMatImage = imread("/home/cesar/CesarBragagnini/MCS/others/wallPaper04.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	imwrite("/home/cesar/CesarBragagnini/MCS/others/dgray.png", inMatImage);
	printf("Imagen %s n de %dx%d\n", "wallPaper04", inMatImage.rows, inMatImage.cols);
	/*
	 *  Sobel Filter
	 */
	serialSobelAsImageIn2d(inMatImage);
	serialSobelAsImageIn1d(inMatImage);
	cudaSobelInLinearMemoryGrid1dBlock1d(inMatImage);
	for(int j = 0; j < 3; j++)
		cudaSobelInLinearMemoryGrid2dBlock2d(inMatImage, width[j]);

    return 0;
}
